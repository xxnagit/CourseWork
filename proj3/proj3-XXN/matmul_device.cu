#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "matmul_device.cuh"

#define BLOCK_WIDTH 32
/*
 * Read TODO items below
 */




__global__
void naiveMatmul(float *a, float *b, float *c, int n) {
    int j = blockIdx.x * blockDim.x + threadIdx.x; //column
    int i = blockIdx.y * blockDim.y + threadIdx.y; //row

    float acc = 0;
    for (int k=0; k<n; k++) {
	acc += a[i*n+k] * b[k*n+j]; //each thread access an element of a and b
    //threads access a in the col order and b in the row order at the same time.
    }
    c[i*n+j] = acc;
}


        

__global__
void cacheMatmul(float *a, float *b, float *c, int n) {
    // TODO: replace this function with cache friendly version
    //Devide the three matrix into sub blocks of size BLOCK_WIDTH * BLOCK_WIDTH;
    //The inner loop calculates two sub blocks of a and b of size BLOCK_WIDTH * BLOCK_WIDTH;
    //In the outer loop, each sub block C needs one row of blocks of a and one column of blocks
    //of b.
    
    
    float acc = 0;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
 

    for (int i = 0; i < (n/BLOCK_WIDTH); i++) {
        for (int j = 0; j < BLOCK_WIDTH; ++j)
            acc += a[row*n + i*BLOCK_WIDTH + j] * b[(i*BLOCK_WIDTH + j)*n + col];
    }

    c[(blockIdx.y * blockDim.y + threadIdx.y)*n+(blockIdx.x*blockDim.x)+threadIdx.x]= acc;
    
    
    //Threads access a in the row order and b to get c in the row order synchronisedly.
    //The entire a should be access n times in order to get the entire c
    //Each cloumns of thread do multiplication with one a[i, row] * b[row,*] = c[i,*];
    //For all threads within each iteration, it scans onw row of a and entire b to get one row.
    //Although in this way it synchronises access by row, the b is access n times that causes
    //the entire calculation every slow.
    /*
    int row = blockIdx.y * blockDim.y + threadIdx.y; //i
    int col = blockIdx.x * blockDim.x + threadIdx.x; //k
    
    
    for(int i = 0; i < n; i++) {
        float r = b[row * n + col];
        atomicAdd(&c[i * n + col], r * a[i *n + row]);
    }
    __syncthreads();
    */
  
}

__global__
void sharedMatmul(float *a, float *b, float *c, int n) {
    // TODO: replace this function with optimized code using
    // shared memory
    

    __shared__ float sharedMemA[BLOCK_WIDTH][BLOCK_WIDTH];
    __shared__ float sharedMemB[BLOCK_WIDTH][BLOCK_WIDTH];

    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    float acc = 0;

    for (int i = 0; i < n/BLOCK_WIDTH; i++) {
        sharedMemA[threadIdx.y][threadIdx.x] = a[row * n + (i * BLOCK_WIDTH + threadIdx.x)];
        sharedMemB[threadIdx.y][threadIdx.x] = b[(i * BLOCK_WIDTH + threadIdx.y) * n + col];


        __syncthreads();


        for (int j = 0; j < BLOCK_WIDTH; j++) {
            acc += sharedMemA[threadIdx.y][j] * sharedMemB[j][threadIdx.x];
        }


        __syncthreads();
    }

    c[row * n + col] = acc;
}

void cudaMatmul(float *a, float *b, float *c, int n, MatmulImplementation type)
{
    // TODO: play with the gridSize and blockSize to find the best one
    if (type == NAIVE) {
        dim3 blockSize(32, 32);
        dim3 gridSize(n / 32, n / 32);
        naiveMatmul<<<gridSize, blockSize>>>(a,b,c,n);
    }
    else if (type == CACHE) {
        dim3 blockSize(32, 32);
        dim3 gridSize(n / 32, n / 32);
        cacheMatmul<<<gridSize, blockSize>>>(a,b,c,n);
    }
    else if (type == SHARED) {
        dim3 blockSize(32, 32);
        dim3 gridSize(n / 32, n / 32);
        sharedMatmul<<<gridSize, blockSize>>>(a,b,c,n);
    }
    // Unknown type
    else
        assert(false);
}
